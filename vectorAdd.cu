#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Includes
#include <stdio.h>
#include <cutil_inline.h>
#include <time.h>

#define REMOTE2
#define GPU
#define ITER 1000

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

// Functions
void Cleanup(void);
void RandomInit(float*, int);
void DataInit(float*, int, const char*);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int n = ITER;
    if (i < N){
      C[i]= 0;
      for (int j = 0; j< n; j++)
      {
        C[i]= (C[i]+A[i]) / B[i];
      }
    }
}

void Host(const float* A, const float* B, float* C, int N)
{
  int n = ITER;
  for (int i =0; i < N; i++)
  {
    if (i < N){
      C[i] = 0;
      for (int j = 0; j< n; j++)
      {
        C[i]= (C[i]+A[i]) / B[i];
      }
    }
  }
}

// Host code
extern "C" 
int calculate(char *data, char *result)
{
#ifdef REMOTE
    int N = 10000;
#else 
    int N = 10000000;
#endif
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();
    
    // Initialize input vectors
#ifdef REMOTE
    ////printf("dane: %s\n", data);
    char *pre;
    pre = strtok(data," ");
    int i =0;
    while (pre != NULL)
    {
      h_A[i]= atof(pre);
      h_B[i]= atof(pre);
      pre = strtok (NULL, " ");
      i ++;
    }
    N = i;
    size = N * sizeof(float);
#else
    DataInit(h_A, N, data);
    DataInit(h_B, N, data);
#endif

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_A, size) );
    cutilSafeCall( hipMalloc((void**)&d_B, size) );
    cutilSafeCall( hipMalloc((void**)&d_C, size) );

    clock_t start = clock();
#ifdef GPU
    // Copy vectors from host memory to device memory
    cutilSafeCall( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    /* Code you want timed here */
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
    cutilSafeCall( hipDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    cutilSafeCall( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    
#else
    Host(h_A, h_B, h_C, N);
#endif
    printf("Time elapsed: %f\n", ((double)clock() - start) / CLOCKS_PER_SEC);

    Cleanup();
    return 0;
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    cutilSafeCall( hipDeviceReset() );
    
    /*if (!noprompt) {
        printf("\nPress ENTER to exit...\n");
        fflush( stdout);
        fflush( stderr);
        getchar();
    }

    exit(0);*/
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void DataInit(float* data, int n, const char* fill)
{
    for (int i = 0; i < n; ++i)
        data[i] = atof(fill);
}
// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i)
        if (strcmp(argv[i], "--noprompt") == 0 ||
			strcmp(argv[i], "-noprompt") == 0) 
		{
            noprompt = true;
            break;
        }
}
