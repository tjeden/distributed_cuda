#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 3
 * of the programming guide with some additions like error checking.
 *
 */

// Includes
#include <stdio.h>
#include <cutil_inline.h>

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
bool noprompt = false;

// Functions
void Cleanup(void);
void RandomInit(float*, int);
void DataInit(float*, int, const char*);
void ParseArguments(int, char**);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Host code
extern "C" 
int calculate(const char *data, char *result)
{
    printf("Vector addition\n");
    printf("dane: %s\n", data);
    int N = 100;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup();
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup();
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup();
    
    // Initialize input vectors
    DataInit(h_A, N, data);
    DataInit(h_B, N, data);

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_A, size) );
    cutilSafeCall( hipMalloc((void**)&d_B, size) );
    cutilSafeCall( hipMalloc((void**)&d_C, size) );

    // Copy vectors from host memory to device memory
    cutilSafeCall( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
    cutilSafeCall( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    cutilCheckMsg("kernel launch failure");
#ifdef _DEBUG
    cutilSafeCall( hipDeviceSynchronize() );
#endif

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    cutilSafeCall( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
    
    // Verify result
    int i;
    for (i = 0; i < N; ++i) {
        //printf("A: %f B: %f C: %f\n", h_A[i], h_B[i], h_C[i]);
        float sum = h_A[i] + h_B[i];
        //result[i] = h_C[0];//(char *)h_C;
        if (fabs(h_C[i] - sum) > 1e-5)
            break;
    }
        sprintf(result,"%f",h_C[0]);
    printf("%s \n", (i == N) ? "PASSED" : "FAILED");

    
    Cleanup();
    return 0;
}

void Cleanup(void)
{
    // Free device memory
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    cutilSafeCall( hipDeviceReset() );
    
    /*if (!noprompt) {
        printf("\nPress ENTER to exit...\n");
        fflush( stdout);
        fflush( stderr);
        getchar();
    }

    exit(0);*/
}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void DataInit(float* data, int n, const char* fill)
{
    for (int i = 0; i < n; ++i)
        data[i] = atof(fill);
}
// Parse program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i)
        if (strcmp(argv[i], "--noprompt") == 0 ||
			strcmp(argv[i], "-noprompt") == 0) 
		{
            noprompt = true;
            break;
        }
}
